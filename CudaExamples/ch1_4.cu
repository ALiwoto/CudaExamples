#include <hip/hip_runtime.h>


#include <hip/hip_runtime_api.h>
#include <stdio.h>

// Maximum threads per block for this example
const int THREADS_PER_BLOCK = 256;

__global__ void sumArray_ch1_4(
    int* inputArray,
    int* blockSums,
    int arraySize
) {
    // Shared memory for this thread block
    __shared__ int sharedData[THREADS_PER_BLOCK];

    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int localThreadId = threadIdx.x;

    // Load data into shared memory
    if (threadId < arraySize) {
        sharedData[localThreadId] = inputArray[threadId];
    }
    else {
        sharedData[localThreadId] = 0;
    }

    // Wait for all threads to load their data
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (localThreadId < stride) {
            sharedData[localThreadId] += sharedData[localThreadId + stride];
        }
        __syncthreads();
    }

    // First thread in block writes result
    if (localThreadId == 0) {
        blockSums[blockIdx.x] = sharedData[0];
        printf("Block %d sum: %d\n", blockIdx.x, sharedData[0]);
    }
}

int ch1_4_main() {
    const int ARRAY_SIZE = 1000;
    const int BYTES_NEEDED = ARRAY_SIZE * sizeof(int);

    // Host arrays
    int* hostInput = new int[ARRAY_SIZE];

    // Initialize input array with numbers 1 to ARRAY_SIZE
    for (int i = 0; i < ARRAY_SIZE; i++) {
        hostInput[i] = 1;  // All ones for easy verification
    }

    // Calculate grid dimensions
    const int NUM_BLOCKS = (ARRAY_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int* hostBlockSums = new int[NUM_BLOCKS];

    // Device arrays
    int* deviceInput, * deviceBlockSums;

    // Allocate GPU memory
    hipMalloc(&deviceInput, BYTES_NEEDED);
    hipMalloc(&deviceBlockSums, NUM_BLOCKS * sizeof(int));

    // Copy data to GPU
    hipMemcpy(deviceInput, hostInput, BYTES_NEEDED, hipMemcpyHostToDevice);

    // Launch kernel
    sumArray_ch1_4<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
        deviceInput,
        deviceBlockSums,
        ARRAY_SIZE
        );

    // Get results back
    hipMemcpy(hostBlockSums, deviceBlockSums, NUM_BLOCKS * sizeof(int), hipMemcpyDeviceToHost);

    // Sum up block sums on CPU
    int totalSum = 0;
    for (int i = 0; i < NUM_BLOCKS; i++) {
        totalSum += hostBlockSums[i];
    }

    printf("\nTotal sum: %d\n", totalSum);
    printf("Expected sum: %d\n", ARRAY_SIZE);  // Should match since all inputs are 1

    // Cleanup
    delete[] hostInput;
    delete[] hostBlockSums;
    hipFree(deviceInput);
    hipFree(deviceBlockSums);

    return 0;
}