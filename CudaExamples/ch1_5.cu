#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cassert>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            printf("CUDA Error: %s at %s:%d\n", \
                   hipGetErrorString(error), \
                   __FILE__, __LINE__); \
            exit(1); \
        } \
    } while(0)

__global__ void debugExample(int* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Print debugging info
    printf("Thread [%d,%d] (global %d) starting...\n",
        blockIdx.x, threadIdx.x, idx);

    // Assert to catch out-of-bounds access
    assert(idx < size);

    // Do some work
    if (idx < size) {
        data[idx] *= 2;
        printf("Thread %d: processed %d -> %d\n",
            idx, data[idx] / 2, data[idx]);
    }
}

int ch1_5_main() {
    const int SIZE = 10;
    int* hostData, * deviceData;

    // Allocate host memory
    hostData = new int[SIZE];
    for (int i = 0; i < SIZE; i++) {
        hostData[i] = i;
    }

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&deviceData, SIZE * sizeof(int)));

    // Copy to device
    CUDA_CHECK(hipMemcpy(deviceData, hostData,
        SIZE * sizeof(int),
        hipMemcpyHostToDevice));

    // Launch kernel with error checking
    debugExample << <2, 5 >> > (deviceData, SIZE);
    CUDA_CHECK(hipGetLastError());  // Check for launch errors
    CUDA_CHECK(hipDeviceSynchronize());  // Check for runtime errors

    // Copy back results
    CUDA_CHECK(hipMemcpy(hostData, deviceData,
        SIZE * sizeof(int),
        hipMemcpyDeviceToHost));

    // Verify results
    printf("\nFinal results:\n");
    for (int i = 0; i < SIZE; i++) {
        printf("hostData[%d] = %d\n", i, hostData[i]);
    }

    // Cleanup
    delete[] hostData;
    CUDA_CHECK(hipFree(deviceData));

    return 0;
}